#include "hip/hip_runtime.h"
// NOT WORKING

#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

__global__ void countOccurrences(char* str, char* word, int* result, int length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int wordLength = strlen(word);

    if (tid < length) {
        if (tid + wordLength <= length) {
            bool match = true;
            for (int i = 0; i < wordLength; i++) {
                if (str[tid + i] != word[i]) {
                    match = false;
                    break;
                }
            }

            if (match) {
                atomicAdd(result, 1);
            }
        }
    }
}

int main() {
    std::string text = "Hello world, hello world!";
    std::string word = "world";
    int textLength = text.size();
    int wordLength = word.size();

    char* d_text;
    char* d_word;
    int* d_result;

    hipMalloc((void**)&d_text, textLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_text, text.c_str(), textLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word.c_str(), wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(d_result, 0, sizeof(int));

    int block_size = 32;
    int num_blocks = (textLength + block_size - 1) / block_size;

    countOccurrences<<<num_blocks, block_size>>>(d_text, d_word, d_result, textLength);

    int result;
    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Occurrences of the word \"" << word << "\": " << result << std::endl;

    hipFree(d_text);
    hipFree(d_word);
    hipFree(d_result);

    return 0;
}
