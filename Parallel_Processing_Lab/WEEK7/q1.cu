#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void adda(int *a, int *b, int *c)
{
    int i = blockIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void addb(int *a, int *b, int *c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int a[20],b[20],c[20];
    int *da, *db, *dc;
    int i, n;

    printf("Enter size of array: ");
    scanf("%d", &n);
    printf("Enter elements of a and b: ");
    for(i=0; i<n*2; i++)
    {
        if (i<n)
            scanf("%d",&a[i]);
        else
            scanf("%d",&b[i-n]);
    }

    int size = sizeof(int)*n;
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);

    adda<<<n,1>>>(da, db, dc);
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
    printf("a) n blocks -> Result: ");
    for(i=0; i<n; i++)
        printf("%d ", c[i]);

    for(i=0;i<n;i++)
        c[i] = 0;
    addb<<<1,n>>>(da, db, dc);
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
    printf("\nb) n threads -> Result: ");
    for(i=0; i<n; i++)
        printf("%d ", c[i]);
    printf("\n");
    
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;

}