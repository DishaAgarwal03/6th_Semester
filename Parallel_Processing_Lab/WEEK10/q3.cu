

#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void power(int *a, int *t) {
    int r = blockDim.x;
    int c = blockDim.y;

    int p = 1, rev = 0, d;
    int ridx = threadIdx.x;
    int cidx = threadIdx.y;

    int i = ridx*c+cidx;
    if(ridx<r-1 && ridx>0 && cidx<c-1 && cidx>0)
    {
        int n = a[i];
        while (n>0)
        {
            d = n%2;
            d = (d==1)?0:1;
            rev += p*d;
            p *= 10;
            n /= 2;
        }
        t[i] = rev;
    }
    else 
        t[i] = a[i];
  
}

void display(int r, int c, int *t)
{
    int i, j;
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
            printf("%d\t",t[i*c+j]);
        printf("\n");
    }
}

int main() 
{
    int *a, *t;
    int r, c, i;
    int *da, *dt;

    printf("Enter number of rows and columns: ");
    scanf("%d %d", &r, &c);

    int size = sizeof(int)*r*c;

    a = (int*)malloc(size);
    t = (int*)malloc(size);

    printf("Enter the elements of the matrix: \n");
    for(i=0;i<r*c;i++)
        scanf("%d", &a[i]);


    hipMalloc((void**)&da, size);
    hipMalloc((void**)&dt, size);
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dt, t, size, hipMemcpyHostToDevice);

    // ELEMENT-WISE
    dim3 gd(1,1,1);
    dim3 bd(r,c,1); // writing this directly in the next statement is not working!!!!
    power<<<gd, bd>>>(da, dt);

    hipMemcpy(t, dt, size, hipMemcpyDeviceToHost);
    printf("\nResult:\n");
    display(r,c,t);

    hipFree(dt);
    hipFree(da);    
    return 0;
}
