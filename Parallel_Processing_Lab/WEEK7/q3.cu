#include "hip/hip_runtime.h"

#include <stdio.h>

#define t 5

__global__ void add(int *a, int *m, int *c, int n, int nm)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = nm / 2;
  
    int sum = 0;
    for(int j = 0; j < nm; j++)
    {
        if(i + j - mid >= 0 && i + j - mid < n)
        {
            sum += a[i + j - mid] * m[j];
        }
    }

    c[i] = sum;
}

int main(void)
{
    int a[20], m[20], c[20];
    int *da, *dm, *dc;
    int n, nm;

    printf("Enter size of array and mask: ");
    scanf("%d %d", &n, &nm);
    printf("Enter elements of a and mask: \n");
    for(int i = 0; i < n + nm; i++)
    {
        if (i < n)
            scanf("%d", &a[i]);
        else
            scanf("%d", &m[i - n]);
    }

    int size_a = sizeof(int) * n;
    int size_m = sizeof(int) * nm;

    hipMalloc((void**)&da, size_a);
    hipMalloc((void**)&dm, size_m);
    hipMalloc((void**)&dc, size_a);

    hipMemcpy(da, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(dm, m, size_m, hipMemcpyHostToDevice);

    int block_size = t;
    int grid_size = (n + block_size - 1) / block_size;

    add<<<grid_size, block_size>>>(da, dm, dc, n, nm);

    hipMemcpy(c, dc, size_a, hipMemcpyDeviceToHost);

    printf("Result: ");
    for(int i = 0; i < n; i++)
        printf("%d ", c[i]);
    printf("\n");

    hipFree(da);
    hipFree(dm);
    hipFree(dc);

    return 0;
}
