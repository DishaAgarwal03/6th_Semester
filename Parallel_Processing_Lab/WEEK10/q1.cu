// COMPLETE IT!!

#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024
#define len 100

// __global__ void mult(int *a, int *t, int c) {
//     int r = threadIdx.x;

//     for(int i=0; i<c; i++)
//         t[r*c+i] = pow(a[r*c+i], r+1);   // 3^1 is giving 2
  
// }

void display(int r, int c, int *t)
{
    int i, j;
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
            printf("%d\t",t[i*c+j]);
        printf("\n");
    }
}

void csr(int *m, int r, int c, int *data, int *colidx, int *rowptr, int *drr, int *kk)
{
    int i, j;
    int k=0, dr=0;
    for (i=0; i<r; i++)
    {
        rowptr[dr++]=k;
        for(j=0; j<c; j++)
        {
            int idx = i*c+j;
            if (m[idx]!=0)
            {
                data[k]=m[idx];
                colidx[k++]= c;
            }
        }
    }
    rowptr[dr]=k;
    *drr = dr;
    *kk = k;
}

int main() 
{
    int *a;
    int r, c, i;
    int *da, *dt;
    int dr, k;
    int data[len], rowptr[len], colidx[len];

    printf("Enter number of rows and columns: ");
    scanf("%d %d", &r, &c);

    int size = sizeof(int)*r*c;

    a = (int*)malloc(size);
    // t = (int*)malloc(size);

    printf("Enter the elements of the matrix: \n");
    for(i=0;i<r*c;i++)
        scanf("%d", &a[i]);


    // cudaMalloc((void**)&da, size);
    // cudaMalloc((void**)&dt, size);
    // cudaMemcpy(da, a, size, cudaMemcpyHostToDevice);
    // // cudaMemcpy(dt, t, size, cudaMemcpyHostToDevice);

    // // ROW-WISE
    // power<<<1,r>>>(da, dt, c);

    // cudaMemcpy(a, dt, size, cudaMemcpyDeviceToHost);
    // printf("\nResult:\n");
    // display(r,c,a);

    hipFree(dt);
    hipFree(da);    
    return 0;
}
