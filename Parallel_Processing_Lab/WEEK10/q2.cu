// transpose of matrix

#include "hip/hip_runtime.h"


#include<stdio.h>
#include<stdlib.h>
#include<string.h>

#define N 1024

__global__ void power(int *a, int *t, int c) {
    int r = threadIdx.x;

    for(int i=0; i<c; i++)
        t[r*c+i] = pow(a[r*c+i], r+1);   // 3^1 is giving 2
  
}

void display(int r, int c, int *t)
{
    int i, j;
    for(i=0;i<r;i++)
    {
        for(j=0;j<c;j++)
            printf("%d\t",t[i*c+j]);
        printf("\n");
    }
}

int main() 
{
    int *a;
    int r, c, i;
    int *da, *dt;

    printf("Enter number of rows and columns: ");
    scanf("%d %d", &r, &c);

    int size = sizeof(int)*r*c;

    a = (int*)malloc(size);
    // t = (int*)malloc(size);

    printf("Enter the elements of the matrix: \n");
    for(i=0;i<r*c;i++)
        scanf("%d", &a[i]);


    hipMalloc((void**)&da, size);
    hipMalloc((void**)&dt, size);
    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    // cudaMemcpy(dt, t, size, cudaMemcpyHostToDevice);

    // ROW-WISE
    power<<<1,r>>>(da, dt, c);

    hipMemcpy(a, dt, size, hipMemcpyDeviceToHost);
    printf("\nResult:\n");
    display(r,c,a);

    hipFree(dt);
    hipFree(da);    
    return 0;
}
