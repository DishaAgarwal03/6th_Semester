#include "hip/hip_runtime.h"
#include <math.h>

#include<stdio.h>

__global__ void add(int *a, float *b)
{
    int i = threadIdx.x;
    b[i] = sinf(a[i]);
}

int main(void)
{
    int a[20];
    float b[20];
    int *da;
    float *db;
    int i, n;

    printf("Enter size of array: ");
    scanf("%d", &n);
    printf("Enter %d angles as radians: ",n);
    for(i=0; i<n; i++)
        scanf("%d",&a[i]);

    int size = sizeof(int)*n;
    int sizef = sizeof(float)*n;
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, sizef);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    add<<<1,n>>>(da, db);
    hipMemcpy(b, db, sizef, hipMemcpyDeviceToHost);

    printf("Result: ");
    for(i=0; i<n; i++)
        printf("%f ", b[i]);
    printf("\n");

    hipFree(da);
    hipFree(db);

    return 0;
}