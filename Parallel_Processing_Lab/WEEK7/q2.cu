#include "hip/hip_runtime.h"

#include<stdio.h>
#define t 5

__global__ void add(int *a, int *b, int *c)
{
    int i = blockIdx.x * t + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int a[20],b[20],c[20];
    int *da, *db, *dc;
    int i, n, nb;

    printf("Enter size of array: ");
    scanf("%d", &n);
    printf("Enter elements of a and b: ");
    for(i=0; i<n*2; i++)
    {
        if (i<n)
            scanf("%d",&a[i]);
        else
            scanf("%d",&b[i-n]);
    }

    int size = sizeof(int)*n;
    hipMalloc((void**)&da, size);
    hipMalloc((void**)&db, size);
    hipMalloc((void**)&dc, size);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(db, b, size, hipMemcpyHostToDevice);

    nb = ceil(n/(float)t);
    add<<<nb,t>>>(da, db, dc);
    hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
    printf("Result: ");
    for(i=0; i<n; i++)
        printf("%d ", c[i]);
    printf("\n");
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}